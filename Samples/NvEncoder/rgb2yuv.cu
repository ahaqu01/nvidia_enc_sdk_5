#include "hip/hip_runtime.h"
#include<time.h>
#include<iostream>
#include<opencv/highgui.h>
#include<opencv2/opencv.hpp>
#include<opencv/cxcore.h>
#include<opencv2/highgui/highgui.hpp>
#include<opencv2/imgproc/imgproc.hpp>
# include <stdio.h>
# include <stdlib.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>
using namespace std;
using namespace cv;
#define BLOCK_SIZE 32 //每个块的线程数 32*32
#define PIC_BLOCK 1 //每个线程处理图像块 5*5
//#define GRID_SIZE 16
//const int block_num=480;
//static const int N = 25;

//__device__ int flag;


//超清公式

#define RGB2Y(R, G, B)  ( 16  + 0.183f * (R) + 0.614f * (G) + 0.062f * (B) )
#define RGB2U(R, G, B)  ( 128 - 0.101f * (R) - 0.339f * (G) + 0.439f * (B) )
#define RGB2V(R, G, B)  ( 128 + 0.439f * (R) - 0.399f * (G) - 0.040f * (B) )

/*
#define RGB2Y(R, G, B)  ( 16  + 0.257f * (R) + 0.504f * (G) + 0.098f * (B) )
#define RGB2U(R, G, B)  ( 128 - 0.148f * (R) - 0.291f * (G) + 0.439f * (B) )
#define RGB2V(R, G, B)  ( 128 + 0.439f * (R) - 0.368f * (G) - 0.071f * (B) )
*/

/*
#define RGB2Y(R, G, B)  ( 0.299f * (R) + 0.587f * (G) + 0.114f * (B) )
#define RGB2U(R, G, B)  ( -0.147f * (R) - 0.28886f * (G) + 0.436f * (B) )
#define RGB2V(R, G, B)  ( 0.615f * (R) - 0.51499f * (G) - 0.10001f * (B) )
*/

#define YUV2R(Y, U, V) ( 1.164f *((Y) - 16) + 1.792f * ((V) - 128) )
#define YUV2G(Y, U, V) ( 1.164f *((Y) - 16) - 0.213f *((U) - 128) - 0.534f *((V) - 128) )
#define YUV2B(Y, U, V) ( 1.164f *((Y) - 16) + 2.114f *((U) - 128))

#define CLIPVALUE(x, minValue, maxValue) ((x) < (minValue) ? (minValue) : ((x) > (maxValue) ? (maxValue) : (x)))

__global__ static void __RgbToYuv420p(const unsigned char* dpRgbData, size_t rgbPitch, unsigned char* dpYuv420pData, size_t yuv420Pitch, int width, int height)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
    //for(int index=0;index<width*height;index++)
    //{
    //printf("index=%d\n",index);
	int w = index % yuv420Pitch; //线程对应的RGB图像列
	int h = index / yuv420Pitch; //线程对应的RGB图像行
    
	if (w >= width || h >= height)
		return;
        
    //printf("index=%d\t",index);
    //printf("w=%d,h=%d\n",w,h);
	unsigned char* dp_y_data = dpYuv420pData; //y通道存在前width*height数组中
	unsigned char* dp_u_data = dp_y_data + height * yuv420Pitch;  //yuv420Pitch RGB图像的列长
	unsigned char* dp_v_data = dp_u_data + height * yuv420Pitch / 4;
    //printf("h=%d,w=%d,rgbPitch=%d\t",h,w,rgbPitch);
	unsigned char r = dpRgbData[h * rgbPitch + w * 3 + 0]; //rgbPitch RGB图像的列长
	unsigned char g = dpRgbData[h * rgbPitch + w * 3 + 1];
	unsigned char b = dpRgbData[h * rgbPitch + w * 3 + 2];

	dp_y_data[h   * yuv420Pitch + w] = (unsigned char)(CLIPVALUE(RGB2Y(r, g, b), 0, 255));
	int num = h / 2 * width / 2 + w / 2;
	int offset = num / width * (yuv420Pitch - width);

	if (h % 2 == 0 && w % 2 == 0)
	{
		dp_u_data[num + offset] = (unsigned char)(CLIPVALUE(RGB2U(r, g, b), 0, 255));
		dp_v_data[num + offset] = (unsigned char)(CLIPVALUE(RGB2V(r, g, b), 0, 255));
	}
    
    //printf("in __RgbToYuv420p\n");
    //printf("%d,%d,%d\t",dpYuv420pData[h * yuv420Pitch + w],dp_u_data[num + offset],dp_v_data[num + offset]);
    //printf("[%d,%d,%d]\t",r,g,b);
    //printf("%d\t",dpYuv420pData[h * yuv420Pitch + w]);
    //printf("\n");
    
    #if 0
    if(threadIdx.x==0)
    {
        //printf("dp_y_data=%d,dp_u_data=%d,dp_v_data=%d\n",dp_y_data[h * yuv420Pitch + w],dp_u_data[num + offset],dp_v_data[num + offset]);
        printf("dpYuv420pData[h * yuv420Pitch + w]=%d\n",dpYuv420pData[h * yuv420Pitch + w]);
    }
    #endif
    //}
}


__global__ static void __RgbToNv12(const unsigned char* dpRgbData, size_t rgbPitch, unsigned char* dpNv12Data, size_t nv12Pitch, int width, int height)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int w = index % nv12Pitch;
	int h = index / nv12Pitch;

	if (w >= width || h >= height)
		return;

	unsigned char* dp_y_data = dpNv12Data;
	unsigned char* dp_u_data = dp_y_data + height * nv12Pitch;
	
	unsigned char r = dpRgbData[h * rgbPitch + w * 3 + 0];
	unsigned char g = dpRgbData[h * rgbPitch + w * 3 + 1];
	unsigned char b = dpRgbData[h * rgbPitch + w * 3 + 2];

	dp_y_data[h * nv12Pitch + w] = (unsigned char)CLIPVALUE(RGB2Y(r, g, b), 0, 255);
	int num = (h / 2 * width / 2 + w / 2);
	int offset = (num * 2 + 1) / width * (nv12Pitch - width);

	if (h % 2 == 0 && w % 2 == 0)
	{
		dp_u_data[num * 2 + 0 + offset] = (unsigned char)(CLIPVALUE(RGB2U(r, g, b), 0, 255));
		dp_u_data[num * 2 + 1 + offset] = (unsigned char)(CLIPVALUE(RGB2V(r, g, b), 0, 255));
	}
}

__global__ static void __RgbToYuv422p(const unsigned char* dpRgbData, size_t rgbPitch, unsigned char* dpYuv422pData, size_t yuv422pPitch, int width, int height)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int w = index % yuv422pPitch;
	int h = index / yuv422pPitch;

	if (w >= width || h >= height)
		return;

	unsigned char* dp_y_data = dpYuv422pData;
	unsigned char* dp_u_data = dp_y_data + height * yuv422pPitch;
	unsigned char* dp_v_data = dp_u_data + height / 2 * yuv422pPitch;

	unsigned char r = dpRgbData[h * rgbPitch + w * 3 + 0];
	unsigned char g = dpRgbData[h * rgbPitch + w * 3 + 1];
	unsigned char b = dpRgbData[h * rgbPitch + w * 3 + 2];

	dp_y_data[h * yuv422pPitch + w] = (unsigned char)CLIPVALUE(RGB2Y(r, g, b), 0, 255);
	int num = h * width / 2 + w / 2;
	int offset = num / width * (yuv422pPitch - width);

	if (w % 2 == 0)
	{
		dp_u_data[num + offset] = (unsigned char)(CLIPVALUE(RGB2U(r, g, b), 0, 255));
		dp_v_data[num + offset] = (unsigned char)(CLIPVALUE(RGB2V(r, g, b), 0, 255));
	}
}


__global__ void print_cuda_dst(uchar *cuda_dst,int rows,int cols)
{
    printf("\n");
    for(int i=0;i<rows*cols*3;i++)
    {
        //printf("in cuda\n");
        printf("%d\t",cuda_dst[i]);
    }
    printf("\n");
}

//指针作为参数传入函数时，只是将指针变量中存储的地址值传入函数，
//在函数内改变了形参的地址值并不会对函数外的指针产生影响
void rgb2yuv(cv::Mat& rgb_img,uchar* yuv_img_buff[])
{
	//printf("in CUDA\n");
	//声明变量
	//bgr图像
	uchar* cuda_src = NULL;
	//yuv图像 destination
	uchar* cuda_dst = NULL;


	//分配空间
    int len_src=sizeof(uchar)*rgb_img.rows*rgb_img.cols*3; //RGB图像大小
    int len_dst=sizeof(uchar)*rgb_img.rows*rgb_img.cols*3/2; //YUV图像大小
	hipMalloc((void**)&cuda_src,len_src);
	hipMalloc((void**)&cuda_dst,len_dst);
    
    /*
	//初始化为0
	hipMemset(change, 0, sizeof(float)*bx*by);
    */
    
	//cpu->gpu
	hipMemcpy(cuda_src, rgb_img.data, len_src, hipMemcpyHostToDevice);

    //分块
	//bx*by块，每个块 BLOCK_SIZE*BLOCK_SIZE个线程（32的倍数最好），每个线程负责pic_block*pic_block小块
	int bx = ((rgb_img.cols + BLOCK_SIZE - 1) / BLOCK_SIZE + PIC_BLOCK - 1) / PIC_BLOCK;
	int by = ((rgb_img.rows + BLOCK_SIZE - 1) / BLOCK_SIZE + PIC_BLOCK - 1) / PIC_BLOCK;
    //printf("bx=%d,by=%d\n",bx,by);
	dim3 blocks(bx*by);
	dim3 threads(BLOCK_SIZE*BLOCK_SIZE);
    
    //dim3 blocks(1);
	//dim3 threads(1);
    size_t rgbPitch=3*rgb_img.cols; //记得乘3！！！！！
    size_t yuv420Pitch=rgb_img.cols;
    //printf("rgbPitch=%d\n",rgbPitch);
	__RgbToYuv420p <<<blocks, threads >>> (cuda_src, rgbPitch,cuda_dst,yuv420Pitch,rgb_img.cols, rgb_img.rows);
      //__RgbToYuv422p<<<blocks, threads >>> (cuda_src, rgbPitch,cuda_dst,yuv420Pitch,rgb_img.cols, rgb_img.rows);
	//gpu->cpu
    /*
    YUV420图像的U/V分量在水平和垂直方向上downsample，在水平和垂直方向上的数据都只有Y分量的一半。
    因此总体来说，U/V分量的数据量分别只有Y分量的1/4，不能作为Mat类型的一个channel。
    所以通常YUV420图像的全部数据存储在Mat的一个channel，比如CV_8UC1，这样对于Mat来说，
    图像的大小就有变化。对于MxN（rows x cols，M行N列）的BGR图像（CV_8UC3);
    其对应的YUV420图像大小是(3M/2)xN（CV_8UC1）。
    前MxN个数据是Y分量，后(M/2)xN个数据是U/V分量，UV数据各占一半。
    */
    
#if 1
    int y_len=sizeof(uchar)*rgb_img.rows*rgb_img.cols;
    int u_len=sizeof(uchar)*rgb_img.rows/2*rgb_img.cols/2;
    int v_len=sizeof(uchar)*rgb_img.rows/2*rgb_img.cols/2;
    hipMemcpy(yuv_img_buff[0], cuda_dst, y_len, hipMemcpyDeviceToHost);
    hipMemcpy(yuv_img_buff[1], cuda_dst+y_len, u_len, hipMemcpyDeviceToHost);
    hipMemcpy(yuv_img_buff[2], cuda_dst+y_len+u_len, v_len, hipMemcpyDeviceToHost);
    
    
    //print_cuda_dst<<<1,1>>>(cuda_dst,rgb_img.rows,rgb_img.cols);
    //print_cuda_dst<<<1,1>>>(cuda_src,rgb_img.rows,rgb_img.cols);
#endif
    
    
#if 0
    Mat yuv_img = Mat::zeros(rgb_img.rows*3/2, rgb_img.cols, CV_8UC1);
	hipMemcpy(yuv_img.data, cuda_dst, len_dst, hipMemcpyDeviceToHost);
    //printf("sizeof(uchar)*rgb_img->rows*rgb_img->cols*3/2=%d\n",sizeof(uchar)*rgb_img->rows*rgb_img->cols*3/2);
    printf("-----------------------");
    //yuv2BGR
    Mat rgbimg(rgb_img.rows,rgb_img.cols,CV_8UC3);
    cvtColor(yuv_img,rgbimg,CV_YUV420p2RGB);
    imwrite("yuv.jpg",rgbimg);
    
    //print_cuda_dst<<<1,1>>>(cuda_src);
    /*
    for(int i=0;i<100;i++)
    {
        for (int j=0;j<1;j++)
        {
            printf("%d\t",yuv_img.data[i]);
        }
        printf("\n");
    }
    */
#endif
	//free
	//printf("int cuda free\n");
	hipFree(cuda_src);
	hipFree(cuda_dst);

}

